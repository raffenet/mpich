#include "hip/hip_runtime.h"
/*
 *  Copyright (C) by Argonne National Laboratory.
 *      See COPYRIGHT in top-level directory.
 */

#include "mpl_gpu_cuda.h"
#include <stdio.h>

#define FASTBOX_SIZE 16*1024*1024

__global__ void MPL_gpu_kernel_trigger(MPL_gpu_event_t *var)
{
    *var -= 1;
    __threadfence_system();
}

__global__ void MPL_gpu_kernel_wait(MPL_gpu_event_t *var)
{
    while(*var > 0);
}

__global__ void MPL_gpu_kernel_fbox_send(void *fastbox, const void *src, size_t len)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    char *dstbuf = (char *)fastbox + 64;
    char *srcbuf = (char *)src;
    volatile int *flag = (int *)fastbox;

    if (idx == 0) {
        while (*flag != 0); /* wait for empty fastbox, mark as busy */
    }
    __syncthreads();

    for (int i = idx; i < len; i += 256) {
       dstbuf[i] = srcbuf[i];
    }

    __threadfence_system();
    __syncthreads();
    if (idx == 0) {
        *flag = 1;
    }
}

__global__ void MPL_gpu_kernel_fbox_recv(void *dst, const void *fastbox, size_t len)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    char *dstbuf = (char *)dst;
    char *srcbuf = (char *)fastbox + 64;
    volatile int *flag = (int *)fastbox;

    if (idx == 0) {
        while (*flag != 1); /* wait for full fastbox */
    }
    __syncthreads();

    for (int i = idx; i < len; i += 256) {
        dstbuf[i] = srcbuf[i];
    }

    __syncthreads();
    if (idx == 0) {
        *flag = 0; /* mark fastbox as empty (0) */
    }
}

extern "C"
void MPL_gpu_enqueue_trigger(volatile int *var, hipStream_t stream)
{
    hipError_t cerr;
    void *args[] = {&var};
    cerr = hipLaunchKernel((const void *) MPL_gpu_kernel_trigger, dim3(1,1,1), dim3(1,1,1),
                            args, 0, stream);
    if (cerr != hipSuccess) {
        fprintf(stderr, "CUDA Error (%s): %s\n", __func__, hipGetErrorString(cerr));
    }
}

extern "C"
void MPL_gpu_enqueue_wait(volatile int *var, hipStream_t stream)
{
    hipError_t cerr;

    void *args[] = {&var};
    cerr = hipLaunchKernel((const void *) MPL_gpu_kernel_wait, dim3(1,1,1), dim3(1,1,1),
                            args, 0, stream);
    if (cerr != hipSuccess) {
        fprintf(stderr, "CUDA Error (%s): %s\n", __func__, hipGetErrorString(cerr));
    }
}

extern "C"
void MPL_gpu_event_init_count(MPL_gpu_event_t *var, int count)
{
    *var = count;
}

extern "C"
void MPL_gpu_event_complete(MPL_gpu_event_t *var)
{
    *var -= 1;
}

extern "C"
bool MPL_gpu_event_is_complete(MPL_gpu_event_t *var)
{
    return (*var) <= 0;
}

extern "C"
void MPL_gpu_send_fastbox(const void *sendbuf, void *fastbox, size_t len, hipStream_t stream)
{
    //printf("sending %lu bytes to %p\n", len, fastbox);
    MPL_gpu_kernel_fbox_send<<<1, 256, 0, stream>>>(fastbox, sendbuf, len);
}

extern "C"
void MPL_gpu_recv_fastbox(const void *fastbox, void *recvbuf, size_t len, hipStream_t stream)
{
    //printf("recving %lu bytes from %p\n", len, fastbox);
    MPL_gpu_kernel_fbox_recv<<<1, 256, 0, stream>>>(recvbuf, fastbox, len);
}

__global__ void MPL_gpu_kernel_copy(void *dst, const void *src, size_t len)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    char *dstbuf = (char *)dst;
    char *srcbuf = (char *)src;

    for (int i = idx; i < len; i += 256) {
       dstbuf[i] = srcbuf[i];
    }
}

extern "C"
void MPL_gpu_memcpy(void *dst, const void *src, size_t len, hipStream_t stream)
{
    MPL_gpu_kernel_copy<<<1, 256, 0, stream>>>(dst, src, len);
}

__global__ void MPL_gpu_kernel_wait(volatile int *flag, int val)
{
    while (*flag != val);
}

__global__ void MPL_gpu_kernel_set(volatile int *flag, int val)
{
    *flag = val;
}

extern "C"
void MPL_gpu_wait_cts(volatile int *flag, hipStream_t stream)
{
    MPL_gpu_kernel_wait<<<1, 1, 0, stream>>>(flag, 0);
}

extern "C"
void MPL_gpu_wait_data(volatile int *flag, hipStream_t stream)
{
    MPL_gpu_kernel_wait<<<1, 1, 0, stream>>>(flag, 1);
}

extern "C"
void MPL_gpu_set_cts(volatile int *flag, hipStream_t stream)
{
    MPL_gpu_kernel_set<<<1, 1, 0, stream>>>(flag, 0);
}

extern "C"
void MPL_gpu_set_data(volatile int *flag, hipStream_t stream)
{
    MPL_gpu_kernel_set<<<1, 1, 0, stream>>>(flag, 1);
}
